/**

Andrew Atkinson
3229071
Assignment 3 Question 1

**/


#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>
#include <time.h>

#define TPB 1

__global__ void findArea( int *a, double *h, double *area, double *y, double *n){

		int index = threadIdx.x + blockIdx.x * blockDim.x;
		if( index < *n){
			y[index] = (*a) + index * (*h);
			area[index] = 1 + y[index] + sin(2*y[index]);
		}
}

int main( int argc, char *argv[]){

	clock_t begin, end;
	double time_spent;

	begin = clock();

	int a = atoi(argv[1]), b = atoi(argv[2]), i, N;
	double h;
	sscanf(argv[3], "%lf", &h);
	double n = (b-a)/h;
	double area = 0;
	N = n/TPB;

	int doubleSize = N*TPB*sizeof(double);

	int *d_a;
	double *d_h, *d_area, *d_x, *d_n;
	double * newArea = (double *)malloc(doubleSize);

	hipMalloc((void **) &d_a, sizeof(int));

	hipMalloc((void **) &d_h, sizeof(double));
	hipMalloc((void **) &d_area, doubleSize);
	hipMalloc((void **) &d_x, doubleSize);
	hipMalloc((void **) &d_n, sizeof(double));

	hipMemcpy(d_a, &a, sizeof(int), hipMemcpyHostToDevice);

	hipMemcpy(d_h, &h, sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(d_n, &n, sizeof(double), hipMemcpyHostToDevice);
	
	findArea<<<N,TPB>>>( d_a, d_h, d_area, d_x, d_n);

	hipMemcpy(newArea, d_area, doubleSize, hipMemcpyDeviceToHost);

	for(i = 0; i < N*TPB; i++){
		area = area + newArea[i];
	}

	area = area*h;

	end = clock();
	time_spent = (double)(end - begin) / CLOCKS_PER_SEC;

	printf("\narea = %lf\na = %d, b = %d, h = 1/%lf\nexecution time = %lf\n", area, a, b, 1/h, time_spent);

	hipFree(d_a);
	hipFree(d_h);
	hipFree(d_area);
	hipFree(d_x);
	hipFree(d_n);
}
